#include "hip/hip_runtime.h"
// cudaBoids.cu - Devon McKee, 2022

#include <glad.h>
#include <GLFW/glfw3.h>
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include ""
#include <vector>
#include "VecMat.h"
#include "Camera.h"
#include "CameraControls.h"
#include "Misc.h"
#include "GLXtras.h"
#include "GeomUtils.h"
#include "dCube.h"

#define cudaCheck(error) if (error != hipSuccess) { printf("CUDA error: %s at %s:%d\n", hipGetErrorString(error), __FILE__, __LINE__); exit(1); }

using std::vector;
using std::string;

GLuint renderProgram = 0;
GLuint boidBuffer = 0;
hipGraphicsResource* boidBufferGraphRes;

// strange issue running more than 1259 boids, needs to be investigated further
const int STARTING_BOIDS = 1259;
const int POINT_SIZE = 2;
const int NUM_GPU_THREADS = 256;

// ----- Simulation constants -----
#define BOID_SPEED 0.005f
#define BOID_PERCEPTION 0.1f
#define WALL_RANGE 0.05f
#define ALIGNMENT_WEIGHT 1.0f
#define COHESION_WEIGHT 1.0f
#define SEPARATION_WEIGHT 1.0f

int win_width = 800, win_height = 800;
Camera camera((float)win_width / win_height, vec3(0, 0, 0), vec3(0, 0, -5));
GLFWwindow* window;
vec3 lightPos = vec3(1, 1, 0);
dCube cube;
hipError_t c_stat;
const char* render_glsl_version = "#version 430";

float bgColor[4] = { 0.5f, 0.5f, 0.5f, 1.0f };
float boidColor[4] = { 0.0f, 0.0f, 0.0f, 1.0f };

// ----- Float3 operators -----
__host__ __device__ float3 operator-(const float3 & v) { return float3{ -v.x, -v.y, -v.z }; }
__host__ __device__ float3 operator+(const float3 & l, const float3 & r) { return float3{ l.x + r.x, l.y + r.y, l.z + r.z }; }
__host__ __device__ float3 operator-(const float3 & l, const float3 & r) { return float3{ l.x - r.x, l.y - r.y, l.z - r.z }; }
__host__ __device__ float3 operator*(const float3 & l, const float3 & r) { return float3{ l.x * r.x, l.y * r.y, l.z * r.z }; }
__host__ __device__ float3 operator*(const float3 & l, float r) { return float3{ l.x * r, l.y * r, l.z * r }; }
__host__ __device__ float3 operator/(const float3 & l, float r) { float _d = 1.f / r; return l * _d; }
__host__ __device__ float b_dist(float3 p1, float3 p2) { return (float)sqrt(pow(p2.x - p1.x, 2) + pow(p2.y - p1.y, 2) + pow(p2.z - p1.z, 2)); }
__host__ __device__ float b_dot(const float3 & a, const float3 & b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
__host__ __device__ float b_length(const float3 & v) { return sqrt(b_dot(v, v)); }
__host__ __device__ float3 b_normalize(const float3 & v) { return v / b_length(v); }

struct Boid;

__global__ void boidKernel(Boid* b, size_t n_boids);

struct Boid {
	float3 pos, vel, col;
	Boid() {
		pos = float3{ rand_float(-1.0f, 1.0f), rand_float(-1.0f, 1.0f), rand_float(-1.0f, 1.0f) };
		vel = b_normalize(float3{ rand_float(-1.0f, 1.0f), rand_float(-1.0f, 1.0f), rand_float(-1.0f, 1.0f) }) * BOID_SPEED;
		col = float3{ boidColor[0], boidColor[1], boidColor[2] };
	}
};

Boid* boids;
Boid* boids_dp;
size_t n_boids = STARTING_BOIDS;

void openGLErrorCallback(GLenum source, GLenum type, GLuint id, GLenum severity, GLsizei length, const GLchar* message, const void* userParam) {
	fprintf(stderr, "GL CALLBACK: %s type = 0x%x, severity = 0x%x, message = %s\n", (type == GL_DEBUG_TYPE_ERROR ? "** GL ERROR **" : ""), type, severity, message);
}

void compileShaders() {
	renderProgram = LinkProgramViaFile("shaders/render.vert", "shaders/render.frag");
	if (!renderProgram) {
		fprintf(stderr, "SHADER: Error linking render shader! Exiting...\n");
		exit(1);
	}
}

void b_initialize() {
	cube.loadBuffer();
	boids = new Boid[n_boids];
	size_t b_size = sizeof(Boid) * n_boids;
	glGenBuffers(1, &boidBuffer);
	glBindBuffer(GL_ARRAY_BUFFER, boidBuffer);
	glBufferData(GL_ARRAY_BUFFER, b_size, boids, GL_STATIC_COPY);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	cudaCheck(hipGraphicsGLRegisterBuffer(&boidBufferGraphRes, boidBuffer, hipGraphicsRegisterFlagsNone));
	cudaCheck(hipGraphicsMapResources(1, &boidBufferGraphRes, 0));
	cudaCheck(hipGraphicsResourceGetMappedPointer((void**)&boids_dp, &b_size, boidBufferGraphRes));
}

void b_terminate() {
	cube.unloadBuffer();
	cudaCheck(hipGraphicsUnmapResources(1, &boidBufferGraphRes, 0));
	cudaCheck(hipGraphicsUnregisterResource(boidBufferGraphRes));
	glDeleteBuffers(1, &boidBuffer);
	delete boids;
}

void compute() {
	// Dispatch kernel
	int num_blocks = (int)floor(n_boids / NUM_GPU_THREADS) + (n_boids % NUM_GPU_THREADS == 0 ? 0 : 1);
	//printf("Dispatching CUDA with %d threads and %d blocks\n", NUM_GPU_THREADS, num_blocks);
	boidKernel<<<num_blocks, NUM_GPU_THREADS>>>(boids_dp, n_boids);
	cudaCheck(hipGetLastError());
	cudaCheck(hipDeviceSynchronize());
}

void display() {
	glClearColor(bgColor[0], bgColor[1], bgColor[2], bgColor[3]);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	cube.display(camera);
	// Render boids
	glUseProgram(renderProgram);
	glPointSize(POINT_SIZE);
	glBindBuffer(GL_ARRAY_BUFFER, boidBuffer);
	VertexAttribPointer(renderProgram, "point", 3, sizeof(Boid), (GLvoid*)offsetof(Boid, pos));
	VertexAttribPointer(renderProgram, "color", 3, sizeof(Boid), (GLvoid*)offsetof(Boid, col));
	glUniform4f(0, boidColor[0], boidColor[1], boidColor[2], boidColor[3]);
	SetUniform(renderProgram, "persp", camera.persp);
	SetUniform(renderProgram, "modelview", camera.modelview);
	glDrawArrays(GL_POINTS, 0, (int)n_boids);
	glFlush();
}

int main() {
	srand((int)time(NULL));
	c_stat = hipSetDevice(0);
	if (c_stat != hipSuccess) { printf("No CUDA-capable GPU found! Exiting...\n"); return 1; }
	if (!glfwInit()) return 1;
	window = glfwCreateWindow(win_width, win_height, "Cuda Boids", NULL, NULL);
	if (!window) { glfwTerminate(); return 1; }
	glfwSetWindowPos(window, 100, 100);
	glfwMakeContextCurrent(window);
	gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
	PrintGLErrors();
	compileShaders();
	glfwWindowHint(GLFW_SAMPLES, 4);
	glfwSwapInterval(1);
	InitializeCallbacks(window);
	//glEnable(GL_DEBUG_OUTPUT);
	glDebugMessageCallback(openGLErrorCallback, 0);
	b_initialize();
	double lastFrame = 0, lastSim = 0;
	while (!glfwWindowShouldClose(window)) {
		double now = glfwGetTime();
		double deltaTime = now - lastFrame;
		if ((now - lastSim) >= (1.0 / 60)) {
			compute();
		}
		display();
		glfwPollEvents();
		glfwSwapBuffers(window);
	}
	b_terminate();
	glfwDestroyWindow(window);
	glfwTerminate();
}

// ----- DEVICE CODE -----

__device__ void b_findNeighbors(Boid b, Boid* boids, size_t n_boids, int* nb, size_t &n_nb) {
	for (size_t i = 0; i < n_boids; i++) {
		float d = b_dist(b.pos, boids[i].pos);
		if (&boids[i] != &b && d < BOID_PERCEPTION && d > 0)
			nb[n_nb++] = i;
	}
}

__device__ float3 b_alignment(Boid b, Boid* boids, int* nb, size_t n_nb) {
	float3 cv = { 0.0f };
	int nc = 0;
	for (size_t i = 0; i < n_nb; i++) {
		size_t n = nb[i];
		cv = cv + boids[n].vel;
		nc++;
	}
	if (nc > 0) {
		cv = cv / (float)nc;
		cv = b_normalize(cv);
		return cv;
	} else {
		return float3{ 0.0f };
	}
}

__device__ float3 b_cohesion(Boid b, Boid* boids, int* nb, size_t n_nb) {
	float3 cv{ 0.0f };
	int nc = 0;
	for (size_t i = 0; i < n_nb; i++) {
		size_t n = nb[i];
		cv = cv + boids[n].pos;
		nc++;
	}
	if (nc > 0) {
		cv = cv / (float)nc;
		cv = cv - b.pos;
		cv = b_normalize(cv);
		return cv;
	} else {
		return float3{ 0.0f };
	}
}

__device__ float3 b_separation(Boid b, Boid* boids, int* nb, size_t n_nb) {
	float3 cv{ 0.0f };
	float nc = 0;
	for (size_t i = 0; i < n_nb; i++) {
		size_t n = nb[i];
		float3 iv = b.pos - boids[n].pos;
		iv = b_normalize(iv);
		iv = iv / b_dist(b.pos, boids[n].pos);
		cv = cv + iv;
		nc++;
	}
	if (nc > 0) {
		cv = cv / nc;
		cv = b_normalize(cv);
		return cv;
	} else {
		return float3{ 0.0f };
	}
}

__device__ float3 b_avoidance(Boid &b) {
	if (b.pos.x > 1.0f) b.pos.x = -1.0f;
	if (b.pos.x < -1.0f) b.pos.x = 1.0f;
	if (b.pos.y > 1.0f) b.pos.y = -1.0f;
	if (b.pos.y < -1.0f) b.pos.y = 1.0f;
	if (b.pos.z > 1.0f) b.pos.z = -1.0f;
	if (b.pos.z < -1.0f) b.pos.z = 1.0f;
	if (b.pos.y > 1 - WALL_RANGE) return float3{ 0.0f, 1 / (-1 - b.pos.y), 0.0f }; // top wall
	if (b.pos.y < -1 + WALL_RANGE) return float3{ 0.0f, 1 / (1 - b.pos.y), 0.0f }; // bottom wall
	return float3{ 0.0f };
}

__global__ void boidKernel(Boid* boids, size_t n_boids) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (idx < n_boids) {
		Boid b = boids[idx];
		// Find neighbors of boid
		int* nb = new int[n_boids];
		size_t n_nb = 0;
		b_findNeighbors(b, boids, n_boids, nb, n_nb);
		// Calculate vectors of influence on boid
		float3 a_vec = b_alignment(b, boids, nb, n_nb) * ALIGNMENT_WEIGHT;
		float3 c_vec = b_cohesion(b, boids, nb, n_nb) * COHESION_WEIGHT;
		float3 s_vec = b_separation(b, boids, nb, n_nb) * SEPARATION_WEIGHT;
		float3 w_vec = b_avoidance(b);
		b.vel = b.vel + a_vec + c_vec + s_vec + w_vec;
		b.vel = b_normalize(b.vel);
		b.vel = b.vel * BOID_SPEED;
		b.pos = b.pos + b.vel;
		delete nb;
		float mp = 1 / BOID_SPEED;
		b.col = float3{ (b.vel.x * mp + 1) / 2, (b.vel.y * mp + 1) / 2, (b.vel.z * mp + 1) / 2 };
		boids[idx] = b;
	}
}